#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

extern "C" void saxpy(int,float,float*,float*);
extern "C" void set(int,float,float*);
extern "C" void map(float*, float*, int);

int main(int argc, char **argv)
{
  float *x, *y, *dx, *dy, tmp;
  int n = 1<<20;

  x = (float*) malloc(n*sizeof(float));
  y = (float*) malloc(n*sizeof(float));
  hipMalloc((void**)&dx,(size_t)n*sizeof(float));
  hipMalloc((void**)&dy,(size_t)n*sizeof(float));

  map(x, dx, n*sizeof(float));
  map(y, dy, n*sizeof(float));

  set(n,1.0f,x);
  set(n,0.0f,y);

  saxpy(n, 2.0, x, y);
  hipMemcpy(&tmp,dy,(size_t)sizeof(float),hipMemcpyDeviceToHost);
  printf("%f\n",tmp);
  return 0;
}
