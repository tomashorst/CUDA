
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv)
{
	hipDeviceProp_t deviceProp;

	int deviceCount = 0;
    	hipError_t error_id = hipGetDeviceCount(&deviceCount);


	printf("En este nodo hay %d placas\n\n",deviceCount);
	for(int dev=0;dev<deviceCount;dev++){
	    	hipSetDevice(dev);
    		hipGetDeviceProperties(&deviceProp, dev);
    		printf("Hola!, yo soy [Device %d: \"%s\"], tu acelerador grafico personal\n", dev, deviceProp.name);
	}

	int dev; hipGetDevice(&dev);
	printf("\nle asigno la device %d, que esta desocupada\n", dev);

	return 0;
}
