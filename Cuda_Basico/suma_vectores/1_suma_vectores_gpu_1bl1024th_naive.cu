#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include "gpu_timer.h"

#define SIZE	1024

// kernel
__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
	int i = threadIdx.x;

	if (i < n)
		c[i] = a[i] + b[i];
}

int main(int argc, char **argv)
{
	int N;
	
	if(argc==2) N=atoi(argv[1]);
	else N=SIZE;

	// punteros a memoria de host
	int *a, *b, *c;

	// punteros a memoria de device
	int *d_a, *d_b, *d_c;

	// alocacion memoria de host
	a = (int *)malloc(N*sizeof(int));
	b = (int *)malloc(N*sizeof(int));
	c = (int *)malloc(N*sizeof(int));

	// alocacion memoria de device
	hipMalloc( &d_a, N*sizeof(int));
	hipMalloc( &d_b, N*sizeof(int));
	hipMalloc( &d_c, N*sizeof(int));

	// inicializacion arrays de host
	for( int i = 0; i < N; ++i )
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	// copia de host a device
	hipMemcpy( d_a, a, N*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, N*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_c, c, N*sizeof(int), hipMemcpyHostToDevice );

	// timer para gpu...
	gpu_timer Reloj;
	Reloj.tic();

	// suma paralela en el device
	VectorAdd<<< 1, N >>>(d_a, d_b, d_c, N);
	
	// milisegundos transcurridos
	printf("VectorAdd<<< 1, N >>>, N= %d t= %lf ms\n", N, Reloj.tac());	

	// copia (solo del resultado) del device a host
	hipMemcpy( c, d_c, N*sizeof(int), hipMemcpyDeviceToHost );

	// verificacion del resultado
	for( int i = 0; i < N; ++i){
		//printf("c[%d] = %d\n", i, c[i]);
		assert(c[i]==2*i);
	}

	// liberacion memoria de host
	free(a);
	free(b);
	free(c);

	// liberacion memoria de device
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
