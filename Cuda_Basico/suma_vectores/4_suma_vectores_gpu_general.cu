#include "hip/hip_runtime.h"
// solucion paralela optima...

#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include "gpu_timer.h"

#define SIZE	1024

// kernel
__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
	// indice de thread mapeado a indice de array 
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		c[i] = a[i] + b[i];
}

int main(int argc, char **argv)
{
	int N;
	
	if(argc==2) N=atoi(argv[1]);
	else N=SIZE;

	// punteros a memoria de host
	int *a, *b, *c;

	// punteros a memoria de device
	int *d_a, *d_b, *d_c;

	// alocacion memoria de host
	a = (int *)malloc(N*sizeof(int));
	b = (int *)malloc(N*sizeof(int));
	c = (int *)malloc(N*sizeof(int));

	// alocacion memoria de device
	hipMalloc( &d_a, N*sizeof(int));
	hipMalloc( &d_b, N*sizeof(int));
	hipMalloc( &d_c, N*sizeof(int));

	// inicializacion arrays de host
	for( int i = 0; i < N; ++i )
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	// copia de host a device
	hipMemcpy( d_a, a, N*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, N*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_c, c, N*sizeof(int), hipMemcpyHostToDevice );

	// timer para gpu...
	gpu_timer Reloj;
	Reloj.tic();

	// grilla de threads suficientemente grande...
	dim3 nThreads(256);
	dim3 nBlocks((N + nThreads.x - 1) / nThreads.x);
	// suma paralela en el device
	VectorAdd<<< nBlocks, nThreads >>>(d_a, d_b, d_c, N);
	
	// milisegundos transcurridos
	printf("VectorAdd<<< %d, %d >>>, N= %d t= %lf ms\n", nBlocks.x, nThreads.x, N, Reloj.tac());	

	// copia (solo del resultado) del device a host
	hipMemcpy( c, d_c, N*sizeof(int), hipMemcpyDeviceToHost );

	// verificacion del resultado
	for( int i = 0; i < N; ++i){
		//printf("c[%d] = %d\n", i, c[i]);
		assert(c[i]==2*i);
	}

	// liberacion memoria de host
	free(a);
	free(b);
	free(c);

	// liberacion memoria de device
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
