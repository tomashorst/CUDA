#include "hip/hip_runtime.h"
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include "simple_timer.h"

#include <thrust/system/cuda/vector.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/pair.h>
#include <thrust/functional.h>
#include <thrust/inner_product.h>
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <sstream>
#include <map>
#include <cassert>

#define USE_NVTX
#ifdef USE_NVTX
#include "nvToolsExt.h"

const uint32_t colors[] = { 0xff00ff00, 0xff0000ff, 0xffffff00, 0xffff00ff, 0xff00ffff, 0xffff0000, 0xffffffff };
const int num_colors = sizeof(colors)/sizeof(uint32_t);

#define PUSH_RANGE(name,cid) { \
    int color_id = cid; \
    color_id = color_id%num_colors;\
    nvtxEventAttributes_t eventAttrib = {0}; \
    eventAttrib.version = NVTX_VERSION; \
    eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE; \
    eventAttrib.colorType = NVTX_COLOR_ARGB; \
    eventAttrib.color = colors[color_id]; \
    eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
    eventAttrib.message.ascii = name; \
    nvtxRangePushEx(&eventAttrib); \
}
#define POP_RANGE nvtxRangePop();
#else
#define PUSH_RANGE(name,cid)
#define POP_RANGE
#endif


// This example demonstrates how to intercept calls to get_temporary_buffer
// and return_temporary_buffer to control how Thrust allocates temporary storage
// during algorithms such as thrust::sort. The idea will be to create a simple
// cache of allocations to search when temporary storage is requested. If a hit
// is found in the cache, we quickly return the cached allocation instead of
// resorting to the more expensive thrust::cuda::malloc.
//
// Note: this implementation cached_allocator is not thread-safe. If multiple
// (host) threads use the same cached_allocator then they should gain exclusive
// access to the allocator before accessing its methods.

struct not_my_pointer
{
  not_my_pointer(void* p)
    : message()
  {
    std::stringstream s;
    s << "Pointer `" << p << "` was not allocated by this allocator.";
    message = s.str();
  }

  virtual ~not_my_pointer() {}

  virtual const char* what() const
  {
    return message.c_str();
  }

private:
  std::string message;
};

// A simple allocator for caching hipMalloc allocations.
struct cached_allocator
{
  typedef char value_type;

  cached_allocator() {}

  ~cached_allocator()
  {
    free_all();
  }

  char *allocate(std::ptrdiff_t num_bytes)
  {
    std::cout << "cached_allocator::allocate(): num_bytes == "
              << num_bytes
              << std::endl;

    char *result = 0;

    // Search the cache for a free block.
    free_blocks_type::iterator free_block = free_blocks.find(num_bytes);

    if (free_block != free_blocks.end())
    {
      std::cout << "cached_allocator::allocate(): found a free block"
                << std::endl;

      result = free_block->second;

      // Erase from the `free_blocks` map.
      free_blocks.erase(free_block);
    }
    else
    {
      // No allocation of the right size exists, so create a new one with
      // `thrust::cuda::malloc`.
      try
      {
        std::cout << "cached_allocator::allocate(): allocating new block"
                  << std::endl;

        // Allocate memory and convert the resulting `thrust::cuda::pointer` to
        // a raw pointer.
        result = thrust::cuda::malloc<char>(num_bytes).get();
      }
      catch (std::runtime_error&)
      {
        throw;
      }
    }

    // Insert the allocated pointer into the `allocated_blocks` map.
    allocated_blocks.insert(std::make_pair(result, num_bytes));

    return result;
  }

  void deallocate(char *ptr, size_t)
  {
    std::cout << "cached_allocator::deallocate(): ptr == "
              << reinterpret_cast<void*>(ptr) << std::endl;

    // Erase the allocated block from the allocated blocks map.
    allocated_blocks_type::iterator iter = allocated_blocks.find(ptr);

    if (iter == allocated_blocks.end())
      throw not_my_pointer(reinterpret_cast<void*>(ptr));

    std::ptrdiff_t num_bytes = iter->second;
    allocated_blocks.erase(iter);

    // Insert the block into the free blocks map.
    free_blocks.insert(std::make_pair(num_bytes, ptr));
  }

private:
  typedef std::multimap<std::ptrdiff_t, char*> free_blocks_type;
  typedef std::map<char*, std::ptrdiff_t>      allocated_blocks_type;

  free_blocks_type      free_blocks;
  allocated_blocks_type allocated_blocks;

  void free_all()
  {
    std::cout << "cached_allocator::free_all()" << std::endl;

    // Deallocate all outstanding blocks in both lists.
    for ( free_blocks_type::iterator i = free_blocks.begin()
        ; i != free_blocks.end()
        ; ++i)
    {
      // Transform the pointer to cuda::pointer before calling cuda::free.
      thrust::cuda::free(thrust::cuda::pointer<char>(i->second));
    }

    for( allocated_blocks_type::iterator i = allocated_blocks.begin()
       ; i != allocated_blocks.end()
       ; ++i)
    {
      // Transform the pointer to cuda::pointer before calling cuda::free.
      thrust::cuda::free(thrust::cuda::pointer<char>(i->first));
    }
  }
};

int N;
#define NSTREAMS	3

std::ofstream logout("log.dat");

void test1()
{
	thrust::device_vector<float> output(N);
	thrust::device_vector<float> values(N,(1.0/N));

        cached_allocator alloc;

	cpu_timer reloj;
	reloj.tic();

	auto irrelevante=thrust::make_discard_iterator();

	hipStream_t s[NSTREAMS];
  	for(int i=0;i<NSTREAMS;i++) hipStreamCreate(&s[i]);

	int n;

	for(int i=0;i<N;i++){

		auto keys=thrust::make_constant_iterator(i);
		thrust::reduce_by_key(thrust::cuda::par(alloc).on(s[n]),
			keys,keys+N,values.begin(),
			irrelevante,
			output.begin()+i
		);
		n++;
		n=n%NSTREAMS;		
	}

  	for(int i=0;i<NSTREAMS;i++) hipStreamSynchronize(s[i]);

	// debug check de sumas parciales
	//for(int i=0;i<N;i++)
	//std::cout << output[i] << std::endl;

	logout << "final result 1=" << thrust::reduce(output.begin(),output.end()) << std::endl; 
	logout << "En ms=" << reloj.tac() << std::endl;
}


void test2()
{
	thrust::cuda::vector<float> values(N,(1.0/N));

	cpu_timer reloj;
	reloj.tic();

        cached_allocator alloc;

	float acum=0.0;
	for(int i=0;i<N;i++){
		acum+=thrust::reduce(thrust::cuda::par(alloc),values.begin(),values.end());
	}

	logout << "final result 2=" << acum << std::endl; 
	logout << "En ms=" << reloj.tac() << std::endl;
}

void test3()
{
	thrust::cuda::vector<float> values(N,(1.0/N));

	cpu_timer reloj;
	reloj.tic();

	float acum=0.0;
	for(int i=0;i<N;i++){
		acum+=thrust::reduce(values.begin(),values.end());
	}

	logout << "final result 3=" << acum << std::endl; 
	logout << "En ms=" << reloj.tac() << std::endl;
}


__global__ void kernel4(float *v, float *o,int N)
{
	extern __shared__ float sdata[];

   	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
   	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;

	// indice lineal de hilo en bloque, tid=0,1,...,blockDim.x*blockDim.y
	unsigned int tid = threadIdx.x + blockDim.x*threadIdx.y;

	if(ix<N && iy<N)
	sdata[tid]=v[ix]*v[iy];
	else sdata[tid]=0.0;
	
	__syncthreads();

	// reduccion serial...
	if(tid==0){
		float acum=0.0;	

		// indice linea de bloque en la grilla
		unsigned int bid = blockIdx.x+gridDim.x*blockIdx.y;
		for(int i=0;i<blockDim.x*blockDim.y;i++) acum += sdata[i];
		o[bid]=acum;	
	}
}

void test4()
{
	cpu_timer reloj;
	reloj.tic();

	thrust::device_vector<float> values(N,1.0/N);
	float *v_raw=thrust::raw_pointer_cast(&values[0]);

	size_t nthreads=32;
	size_t nblocks=(N+nthreads-1)/nthreads;

	// 1 block 1024 threads	

	// nthreads*nthreads*nblocks*nblocks >= N^2
	dim3 nt(nthreads,nthreads);
	dim3 nb(nblocks,nblocks);

	thrust::device_vector<float> parciales(nblocks*nblocks);
	float *parciales_raw=thrust::raw_pointer_cast(&parciales[0]);

	size_t smem = nthreads*nthreads*sizeof(float); 

	kernel4<<<nb,nt,smem>>>(v_raw,parciales_raw,N);
	hipDeviceSynchronize();

	float acum= thrust::reduce(parciales.begin(),parciales.end()) ;
	logout << "final result 4=" << acum << std::endl; 
	logout << "En ms=" << reloj.tac() << std::endl;

	for(int i=0;i<nblocks*nblocks;i++)
	std::cout << parciales[i] << " ";
	std::cout << std::endl;

}

void test5()
{
	cpu_timer reloj;
	reloj.tic();

	thrust::device_vector<float> values(N,1.0/N);

	float acum=0.0;
	for(int i=0;i<N;i++){
		float value0=values[i];
		auto const_it=thrust::make_constant_iterator(value0);
		acum+=thrust::inner_product(values.begin(),values.end(),const_it,0.0);
	}

	logout << "final result 5=" << acum << std::endl; 
	logout << "En ms=" << reloj.tac() << std::endl;	
}


int main(int argc, char **argv){

	if(argc>1) N = atoi(argv[1]);
	else N=128;

	PUSH_RANGE("Test1",1)	
	test1();
	POP_RANGE

	PUSH_RANGE("Test2",2)	
	test2();
	POP_RANGE

	PUSH_RANGE("Test3",3)	
	test3();
	POP_RANGE

	PUSH_RANGE("Test4",4)	
	test4();
	POP_RANGE

	PUSH_RANGE("Test4",4)	
	test5();
	POP_RANGE

/*	float acum=0.0;
	for(int i=0;i<N;i++){
	for(int j=0;j<N;j++){
		acum+=(1.0/N)*(1.0/N);
	}}
	std::cout << "check=" << acum << std::endl;
*/
	return 0;
}

