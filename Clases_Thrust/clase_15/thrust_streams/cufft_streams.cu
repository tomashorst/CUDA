
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <hipfft/hipfft.h>

#define NUM_STREAMS 3

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/********/
/* MAIN */
/********/
int main()
{
    const int N = 5000;

    // --- Host input data initialization
    float2 *h_in1 = new float2[N];
    float2 *h_in2 = new float2[N];
    float2 *h_in3 = new float2[N];
    for (int i = 0; i < N; i++) {
        h_in1[i].x = 1.f;
        h_in1[i].y = 0.f;
        h_in2[i].x = 1.f;
        h_in2[i].y = 0.f;
        h_in3[i].x = 1.f;
        h_in3[i].y = 0.f;
    }

    // --- Host output data initialization
    float2 *h_out1 = new float2[N];
    float2 *h_out2 = new float2[N];
    float2 *h_out3 = new float2[N];
    for (int i = 0; i < N; i++) {
        h_out1[i].x = 0.f;
        h_out1[i].y = 0.f;
        h_out2[i].x = 0.f;
        h_out2[i].y = 0.f;
        h_out3[i].x = 0.f;
        h_out3[i].y = 0.f;
    }

    // --- Registers host memory as page-locked (required for asynch cudaMemcpyAsync)
    gpuErrchk(hipHostRegister(h_in1, N*sizeof(float2), hipHostRegisterPortable));
    gpuErrchk(hipHostRegister(h_in2, N*sizeof(float2), hipHostRegisterPortable));
    gpuErrchk(hipHostRegister(h_in3, N*sizeof(float2), hipHostRegisterPortable));
    gpuErrchk(hipHostRegister(h_out1, N*sizeof(float2), hipHostRegisterPortable));
    gpuErrchk(hipHostRegister(h_out2, N*sizeof(float2), hipHostRegisterPortable));
    gpuErrchk(hipHostRegister(h_out3, N*sizeof(float2), hipHostRegisterPortable));

    // --- Device input data allocation
    float2 *d_in1;          gpuErrchk(hipMalloc((void**)&d_in1, N*sizeof(float2)));
    float2 *d_in2;          gpuErrchk(hipMalloc((void**)&d_in2, N*sizeof(float2)));
    float2 *d_in3;          gpuErrchk(hipMalloc((void**)&d_in3, N*sizeof(float2)));
    float2 *d_out1;         gpuErrchk(hipMalloc((void**)&d_out1, N*sizeof(float2)));
    float2 *d_out2;         gpuErrchk(hipMalloc((void**)&d_out2, N*sizeof(float2)));
    float2 *d_out3;         gpuErrchk(hipMalloc((void**)&d_out3, N*sizeof(float2)));

    // --- Creates CUDA streams
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) gpuErrchk(hipStreamCreate(&streams[i]));

    // --- Creates cuFFT plans and sets them in streams
    hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*NUM_STREAMS);
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipfftPlan1d(&plans[i], N, HIPFFT_C2C, 1);
        hipfftSetStream(plans[i], streams[i]);
    }

    // --- Async memcopyes and computations
    gpuErrchk(hipMemcpyAsync(d_in1, h_in1, N*sizeof(float2), hipMemcpyHostToDevice, streams[0]));
    gpuErrchk(hipMemcpyAsync(d_in2, h_in2, N*sizeof(float2), hipMemcpyHostToDevice, streams[1]));
    gpuErrchk(hipMemcpyAsync(d_in3, h_in3, N*sizeof(float2), hipMemcpyHostToDevice, streams[2]));
    hipfftExecC2C(plans[0], (hipfftComplex*)d_in1, (hipfftComplex*)d_out1, HIPFFT_FORWARD);
    hipfftExecC2C(plans[1], (hipfftComplex*)d_in2, (hipfftComplex*)d_out2, HIPFFT_FORWARD);
    hipfftExecC2C(plans[2], (hipfftComplex*)d_in3, (hipfftComplex*)d_out3, HIPFFT_FORWARD);
    gpuErrchk(hipMemcpyAsync(h_out1, d_out1, N*sizeof(float2), hipMemcpyDeviceToHost, streams[0]));
    gpuErrchk(hipMemcpyAsync(h_out2, d_out2, N*sizeof(float2), hipMemcpyDeviceToHost, streams[1]));
    gpuErrchk(hipMemcpyAsync(h_out3, d_out3, N*sizeof(float2), hipMemcpyDeviceToHost, streams[2]));

    for(int i = 0; i < NUM_STREAMS; i++)
        gpuErrchk(hipStreamSynchronize(streams[i]));

    // --- Releases resources
    gpuErrchk(hipHostUnregister(h_in1));
    gpuErrchk(hipHostUnregister(h_in2));
    gpuErrchk(hipHostUnregister(h_in3));
    gpuErrchk(hipHostUnregister(h_out1));
    gpuErrchk(hipHostUnregister(h_out2));
    gpuErrchk(hipHostUnregister(h_out3));
    gpuErrchk(hipFree(d_in1));
    gpuErrchk(hipFree(d_in2));
    gpuErrchk(hipFree(d_in3));
    gpuErrchk(hipFree(d_out1));
    gpuErrchk(hipFree(d_out2));
    gpuErrchk(hipFree(d_out3));

    for(int i = 0; i < NUM_STREAMS; i++) gpuErrchk(hipStreamDestroy(streams[i]));

    delete[] h_in1;
    delete[] h_in2;
    delete[] h_in3;
    delete[] h_out1;
    delete[] h_out2;
    delete[] h_out3;

    hipDeviceReset();  

    return 0;
}
