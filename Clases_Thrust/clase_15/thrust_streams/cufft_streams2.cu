// Compile with:
// nvcc --std=c++11 stream_parallel.cu -o stream_parallel -lcufft

#include <iostream>

#include <hip/hip_runtime.h>


#include <hipfft/hipfft.h>

// Print file name, line number, and error code when a CUDA error occurs.
#define check_cuda_errors(val)  __check_cuda_errors__ ( (val), #val, __FILE__, __LINE__ )

template <typename T>
inline void __check_cuda_errors__(T code, const char *func, const char *file, int line) {
    if (code) {
    std::cout << "CUDA error at "
          << file << ":" << line << std::endl
          << "error code: " << (unsigned int) code
          << " type: \""  << hipGetErrorString(hipGetLastError()) << "\"" << std::endl
          << "func: \"" << func << "\""
          << std::endl;
    hipDeviceReset();
    exit(EXIT_FAILURE);
    }
}

int main(int argc, char *argv[]) {

    // Number of FFTs to compute.
    const int NUM_DATA = 64;

    // Length of each FFT.
    const int N = 1048576;

    // Number of GPU streams across which to distribute the FFTs.
    const int NUM_STREAMS = 4;

    // Allocate and initialize host input data.
    float2 **h_in = new float2 *[NUM_STREAMS];
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        h_in[ii] = new float2[N];
        for (int jj = 0; jj < N; ++jj) {
            h_in[ii][jj].x = (float) 1.f;
            h_in[ii][jj].y = (float) 0.f;
        }
    }

    // Allocate and initialize host output data.
    float2 **h_out = new float2 *[NUM_STREAMS];
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
    h_out[ii] = new float2[N];
    for (int jj = 0; jj < N; ++jj) {
            h_out[ii][jj].x = 0.f;
            h_out[ii][jj].y = 0.f;
        }
    }

    // Pin host input and output memory for cudaMemcpyAsync.
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        check_cuda_errors(hipHostRegister(h_in[ii], N*sizeof(float2), hipHostRegisterPortable));
        check_cuda_errors(hipHostRegister(h_out[ii], N*sizeof(float2), hipHostRegisterPortable));
    }

    // Allocate pointers to device input and output arrays.
    float2 **d_in = new float2 *[NUM_STREAMS];
    float2 **d_out = new float2 *[NUM_STREAMS];

    // Allocate intput and output arrays on device.
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        check_cuda_errors(hipMalloc((void**)&d_in[ii], N*sizeof(float2)));
        check_cuda_errors(hipMalloc((void**)&d_out[ii], N*sizeof(float2)));
    }

    // Create CUDA streams.
    hipStream_t streams[NUM_STREAMS];
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        check_cuda_errors(hipStreamCreate(&streams[ii]));
    }

    // Creates cuFFT plans and sets them in streams
    hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*NUM_STREAMS);
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        hipfftPlan1d(&plans[ii], N, HIPFFT_C2C, 1);
        hipfftSetStream(plans[ii], streams[ii]);
    }

    // Fill streams with async memcopies and FFTs.
    for (int ii = 0; ii < NUM_DATA; ii++) {
        int jj = ii % NUM_STREAMS;
        check_cuda_errors(hipMemcpyAsync(d_in[jj], h_in[jj], N*sizeof(float2), hipMemcpyHostToDevice, streams[jj]));
        hipfftExecC2C(plans[jj], (hipfftComplex*)d_in[jj], (hipfftComplex*)d_out[jj], HIPFFT_FORWARD);
        check_cuda_errors(hipMemcpyAsync(h_out[jj], d_out[jj], N*sizeof(float2), hipMemcpyDeviceToHost, streams[jj]));
    }

    // Wait for calculations to complete.
    for(int ii = 0; ii < NUM_STREAMS; ii++) {
        check_cuda_errors(hipStreamSynchronize(streams[ii]));
    }

    // Free memory and streams.
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        check_cuda_errors(hipHostUnregister(h_in[ii]));
        check_cuda_errors(hipHostUnregister(h_out[ii]));
        check_cuda_errors(hipFree(d_in[ii]));
        check_cuda_errors(hipFree(d_out[ii]));
        delete[] h_in[ii];
        delete[] h_out[ii];
        check_cuda_errors(hipStreamDestroy(streams[ii]));
    }

    delete plans;

    hipDeviceReset();  

    return 0;
}
