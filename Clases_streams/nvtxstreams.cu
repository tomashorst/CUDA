// nvcc -I /usr/local/cuda-8.0/targets/x86_64-linux/include nvtxstreams.cu -DUSE_NVTX -lnvToolsExt

#include <hip/hip_runtime.h>
#include <cstdio>

#ifdef USE_NVTX
#include <nvToolsExt.h>

const uint32_t colors[] = { 0xff00ff00, 0xff0000ff, 0xffffff00, 0xffff00ff, 0xff00ffff, 0xffff0000, 0xffffffff };
const int num_colors = sizeof(colors)/sizeof(uint32_t);

#define PUSH_RANGE(name,cid) { \
	int color_id = cid; \
	color_id = color_id%num_colors;\
	nvtxEventAttributes_t eventAttrib = {0}; \
	eventAttrib.version = NVTX_VERSION; \
	eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE; \
	eventAttrib.colorType = NVTX_COLOR_ARGB; \
	eventAttrib.color = colors[color_id]; \
	eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
	eventAttrib.message.ascii = name; \
	nvtxRangePushEx(&eventAttrib); \
}
#define POP_RANGE nvtxRangePop();
#else
#define PUSH_RANGE(name,cid)
#define POP_RANGE
#endif

__global__ void init_data_kernel( int n, double* x)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if ( i < n )
	{
		x[i] = n - i;
	}
}


__global__ void daxpy_kernel(int n, double a, double * x, double * y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)
	{
		y[i] = a*x[i] + y[i];
	}
}

__global__ void check_results_kernel( int n, double correctvalue, double * x )
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)
	{
		if ( x[i] != correctvalue )
		{
			printf("ERROR at index = %d, expected = %f, actual: %f\n",i,correctvalue,x[i]);
		}
	}
}

void init_host_data( int n, double * x )
{
	PUSH_RANGE("init_host_data",1)
	for (int i=0; i<n; ++i)
	{
		x[i] = i;
	}
	POP_RANGE
}

void init_data(int n, double* x, double* x_d, double* y_d)
{
	PUSH_RANGE("init_data",2)
	hipStream_t copy_stream;
	hipStream_t compute_stream;
	hipStreamCreate(&copy_stream);
	hipStreamCreate(&compute_stream);

	hipMemcpyAsync( x_d, x, n*sizeof(double), hipMemcpyDefault, copy_stream );
	init_data_kernel<<<ceil(n/256),256,0,compute_stream>>>(n, y_d);

	hipStreamSynchronize(copy_stream);
	hipStreamSynchronize(compute_stream);

	hipStreamDestroy(compute_stream);
	hipStreamDestroy(copy_stream);
	POP_RANGE
}

void daxpy(int n, double a, double* x_d, double* y_d)
{
	PUSH_RANGE("daxpy",3)
	daxpy_kernel<<<ceil(n/256),256>>>(n,a,x_d,y_d);
	hipDeviceSynchronize();
	POP_RANGE
}

void check_results( int n, double correctvalue, double* x_d )
{
	PUSH_RANGE("check_results",4)
	check_results_kernel<<<ceil(n/256),256>>>(n,correctvalue,x_d);
	POP_RANGE
}

void run_test(int n)
{
	PUSH_RANGE("run_test",0)
	double* x;
	double* x_d;
	double* y_d;
	hipSetDevice(0);
	hipHostMalloc((void**) &x, n*sizeof(double), hipHostMallocDefault);
	hipMalloc((void**)&x_d,n*sizeof(double));
	hipMalloc((void**)&y_d,n*sizeof(double));

	init_host_data(n, x);

	init_data(n,x,x_d,y_d);

	daxpy(n,1.0,x_d,y_d);

	check_results(n, n, y_d);

	hipFree(y_d);
	hipFree(x_d);
	hipHostFree(x);
	hipDeviceSynchronize();
	POP_RANGE
}

int main()
{
	int n = 1<<22;
	run_test(n);
	return 0;
}
