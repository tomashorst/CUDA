/*
  Ejercicios: 
  Timming CPU vs GPU.
  Cambiar y Verificar distribucion.      
*/

/*
 * This program uses the host CURAND API to generate 100 
 * pseudorandom floats.
 */
 #include <stdio.h>
 #include <stdlib.h>
 #include <hip/hip_runtime.h>
 #include <hiprand/hiprand.h>
 
 #define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
     printf("Error at %s:%d\n",__FILE__,__LINE__);\
     return EXIT_FAILURE;}} while(0)
 #define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
     printf("Error at %s:%d\n",__FILE__,__LINE__);\
     return EXIT_FAILURE;}} while(0)
 
 int main(int argc, char *argv[])
 {

     size_t n = 100;

     if(argc>1) n=atoi(argv[1]);

     size_t i;
     hiprandGenerator_t gen;
     float *hostData;
 
     /* Allocate n floats on host */
     hostData = (float *)calloc(n, sizeof(float));
 
    #ifndef CPU
    float *devData; 

     /* Allocate n floats on device */
     CUDA_CALL(hipMalloc((void **)&devData, n*sizeof(float)));
 
     /* Create pseudo-random number generator */
     CURAND_CALL(hiprandCreateGenerator(&gen, 
                 HIPRAND_RNG_PSEUDO_DEFAULT));
     
     /* Set seed */
     CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 
                 1234ULL));
 
     /* Generate n floats on device */
     CURAND_CALL(hiprandGenerateUniform(gen, devData, n));
 
     /* Copy device memory to host */
     CUDA_CALL(hipMemcpy(hostData, devData, n * sizeof(float),
         hipMemcpyDeviceToHost));
    #else    

     /* Create pseudo-random number generator */
     CURAND_CALL(curandCreateGeneratorHost(&gen, 
        CURAND_RNG_PSEUDO_DEFAULT));

     /* Set seed */
     CURAND_CALL(curandSetPseudoRandomGeneratorSeed(gen, 
        1234ULL));

     /* Generate n floats on device */
     CURAND_CALL(curandGenerateUniform(gen, hostData, n));

     #endif

     /* Show result */
     int m=(n>100)?100:n;
     for(i = 0; i < m; i++) {
         printf("%1.4f ", hostData[i]);
     }
     printf("\n");
 
     /* Cleanup */
     CURAND_CALL(hiprandDestroyGenerator(gen));
 
     #ifndef CPU
     CUDA_CALL(hipFree(devData));
     #endif
     free(hostData);    

     return EXIT_SUCCESS;
 }
 